#include <iostream>
#include "dsaX_cublas_interface.h"

using namespace std;

void dsaXHgemmStridedBatchedCuda(half *real_in, half *imag_in, half *real_out, half *imag_out, dsaXBLASParam blas_param) {
#ifdef DSA_XENGINE_TARGET_CUDA
  
  // not sure if essential
  hipDeviceSynchronize();
  
  // Set up for gemm
  hipblasHandle_t cublasH = NULL;
  hipStream_t stream = NULL;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  hipblasCreate(&cublasH);
  hipblasSetStream(cublasH, stream);

  // Transfer params
  hipblasOperation_t transa;
  hipblasOperation_t transb;
  switch (blas_param.trans_a) {
  case DSA_BLAS_OP_N:
    transa = HIPBLAS_OP_N; break;
  case DSA_BLAS_OP_T:
    transa = HIPBLAS_OP_T; break;
  case DSA_BLAS_OP_C:
    transa = HIPBLAS_OP_C; break;
  default:
    std::cout << "Unknown cublas transpose" << std::endl;
  }

  switch (blas_param.trans_b) {
  case DSA_BLAS_OP_N:
    transb = HIPBLAS_OP_N; break;
  case DSA_BLAS_OP_T:
    transb = HIPBLAS_OP_T; break;
  case DSA_BLAS_OP_C:
    transb = HIPBLAS_OP_C; break;
  default:
    std::cout << "Unknown cublas transpose" << std::endl;
  }
  
  const int m = blas_param.m;
  const int n = blas_param.n;
  const int k = blas_param.k;
  const half alpha = blas_param.alpha.real();
  const half malpha = -1.0 * blas_param.alpha.real();
  const int lda = blas_param.lda;
  const int ldb = blas_param.ldb;
  const half beta0 = blas_param.beta.real();
  const half beta1 = 1.0;
  const int ldc = blas_param.ldc;
  const long long int strideA = blas_param.a_stride;
  const long long int strideB = blas_param.b_stride;
  const long long int strideC = blas_param.c_stride;
  const int batchCount = blas_param.batch_count;
  
  // run strided batched gemm for datatype (a + ib)(c + id)
  // ac
  hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			    &alpha,real_in,lda,strideA,
			    real_in,ldb,strideB,&beta0,
			    real_out,ldc,strideC,
			    batchCount);
  // bd
  hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			    &alpha,imag_in,lda,strideA,
			    imag_in,ldb,strideB,&beta1,
			    real_out,ldc,strideC,
			    batchCount);
  // -bc
  hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			    &malpha,imag_in,lda,strideA,
			    real_in,ldb,strideB,&beta0,
			    imag_out,ldc,strideC,
			    batchCount);
  // ad
  hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			    &alpha,real_in,lda,strideA,
			    imag_in,ldb,strideB,&beta1,
			    imag_out,ldc,strideC,
			    batchCount);

  // shown to be essential
  hipDeviceSynchronize();

  // destroy stream
  hipStreamDestroy(stream);
  hipblasDestroy(cublasH);  
#else
  std::cout "Not implemented" << std::endl;
  exit(0);
#endif
}
