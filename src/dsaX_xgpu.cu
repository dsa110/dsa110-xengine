#include "hip/hip_runtime.h"
// -*- c++ -*-
/* will run xgpu */
/* assumes input block size is appropriate */
#include <iostream>
#include <algorithm>
using std::cout;
using std::cerr;
using std::endl;
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <string.h>
#include <unistd.h>
#include <netdb.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netinet/in.h>
#include <time.h>
#include <syslog.h>

#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/scatter.h>

#include "dada_cuda.h"
#include "dada_client.h"
#include "dada_def.h"
#include "dada_hdu.h"
#include "multilog.h"
#include "ipcio.h"
#include "ipcbuf.h"
#include "dada_affinity.h"
#include "ascii_header.h"
#include "dsaX_def.h"
#include "cube/cube.h"
#include "xgpu.h"

/* global variables */
int quit_threads = 0;
char STATE[20];
int DEBUG = 0;

void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out);
int dada_bind_thread_to_core (int core);
void simple_extract (float *matr, float *mati, float *output);

void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out)
{

  if (dada_hdu_unlock_read (in) < 0)
    {
      syslog(LOG_ERR, "could not unlock read on hdu_in");
    }
  dada_hdu_destroy (in);

  if (dada_hdu_unlock_write (out) < 0)
    {
      syslog(LOG_ERR, "could not unlock write on hdu_out");
    }
  dada_hdu_destroy (out);

}

/* functor to make map */
struct map_functor
{

  int n;
  map_functor(int _n) : n(_n) {}
  
  __device__
  int operator()(const int x) const {
    
    int i = (int)(x/(NSNAPS*12));
    return x+i*n;
    
  }
}; 

/* KERNEL */

// input is [Time, Ant (NSNAPS snaps), Chan (NCHANG groups), ant (3 per snap), chan (384 per group), time (2 per packet), pol (2), R/I]
// output is [time, frequency, ANT, pol, ri]
// here, ANT=16, frequency=1536
// strictly expect NNATINTS*NCORRINTS time samples per call. Use NNATINTS/2 blocks and NCORRINTS threads
__global__
void massage(char *inpt, char *output) {

  int idx = blockIdx.x*blockDim.x + threadIdx.x; // global index - runs over Time

  int NBYTES_PER_THREAD = NSNAPS*NCHANG*3*384*2*2; // number of bytes per thread
  int inpt_sidx = NBYTES_PER_THREAD*idx; // start idx for input
  int output_sidx = 2*NCHAN*NSNAPS*3*2*2*idx; // start idx for output
  int inpt_idx,output_idx;

  __shared__ char toutp[2*NCHAN*NSNAPS*3*2*2];
  
  for (int i1=0;i1<NSNAPS;i1++) { // Ant
    for (int i2=0;i2<NCHANG;i2++) { // Chan
      for (int i3=0;i3<3;i3++) { // ant
	for (int i4=0;i4<384;i4++) { // chan
	  for (int i5=0;i5<2;i5++) { // time

	    inpt_idx = inpt_sidx + 2 * (i1*NCHANG*3*384*2 + i2*3*384*2 + i3*384*2 + i4*2 + i5);
	    
	    output_idx = i5*NCHAN*NSNAPS*3*2*2 + (i2*384+i4)*NSNAPS*3*2*2 + (i1*3+i3)*2*2;

            // real parts
	    toutp[output_idx] = ((char)(((unsigned char)(inpt[inpt_idx]) & (unsigned char)(15)) << 4))/16;
	    toutp[output_idx+2] = ((char)(((unsigned char)(inpt[inpt_idx+1]) & (unsigned char)(15)) << 4))/16;
	    // imaginary parts
	    toutp[output_idx+1] = ((char)((unsigned char)(inpt[inpt_idx]) & (unsigned char)(240)))/16;
	    toutp[output_idx+3] = ((char)((unsigned char)(inpt[inpt_idx+1]) & (unsigned char)(240)))/16;	    	    

	  }
	}
      }
    }
  }

  __syncthreads();

  for (int i=0;i<2*NCHAN*NSNAPS*3*2*2;i++) {
    output[output_sidx+i] = toutp[i];
  }
}

void usage()
{
fprintf (stdout,
	   "dsaX_xgpu [options]\n"
	   " -c core   bind process to CPU core [no default]\n"
	   " -d send debug messages to syslog\n"
	   " -h print usage\n");
}

// assumes TRIANGULAR_ORDER for mat (f, baseline, pol, ri)
// based on xGPU xgpuExtractMatrix in cpu_util.c
// output order is (baseline, frequency, pol, r/i)
void simple_extract(float *matr, float *mati, float *output) {

  int in_idx, out_idx;
  for (int f=0;f<NCHAN;f++) {
    for (int bctr=0;bctr<NBASE;bctr++) {
      for (int pol1=0;pol1<NPOL;pol1++) {
      
	in_idx = (f*NBASE+bctr)*4+pol1*3;
	out_idx = 2*((bctr*NCHAN+f)*2+pol1);
	
	output[out_idx] = matr[in_idx];
	output[out_idx+1] = mati[in_idx];
	
      }
    }
  }

}


// MAIN

int main (int argc, char *argv[]) {

// startup syslog message
  // using LOG_LOCAL0
  openlog ("dsaX_xgpu", LOG_CONS | LOG_PID | LOG_NDELAY, LOG_LOCAL0);
  syslog (LOG_NOTICE, "Program started by User %d", getuid ());
  
  /* DADA Header plus Data Unit */
  dada_hdu_t* hdu_in = 0;
  dada_hdu_t* hdu_out = 0;

  // data block HDU keys
  key_t in_key = CAPTURE_BLOCK_KEY;
  key_t out_key = XGPU_BLOCK_KEY;
  
  // command line arguments
  int core = -1;
  int arg = 0;
  
  while ((arg=getopt(argc,argv,"c:dh")) != -1)
    {
      switch (arg)
	{
	case 'c':
	  if (optarg)
	    {
	      core = atoi(optarg);
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-c flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'd':
	  DEBUG=1;
	  syslog (LOG_DEBUG, "Will excrete all debug messages");
	  break;
	case 'h':
	  usage();
	  return EXIT_SUCCESS;
	}
    }

  // record STATE info
  sprintf(STATE,"NOBUFFER");

  // Bind to cpu core
  if (core >= 0)
    {
      if (dada_bind_thread_to_core(core) < 0)
	syslog(LOG_ERR,"failed to bind to core %d", core);
      syslog(LOG_NOTICE,"bound to core %d", core);
    }  
  
  // for scatter operation
  syslog(LOG_INFO, "preparing map");
  uint64_t block_feed = NREORDERS*NNATINTS*NCORRINTS*NCHAN*NSNAPS*3*2*2;
  thrust::device_vector<int> map(block_feed/NREORDERS);
  thrust::sequence(map.begin(),map.end(),(int)0,(int)1);
  int nnn = NANT*2*2-NSNAPS*3*2*2;
  thrust::transform(map.begin(),map.end(),map.begin(),map_functor(nnn));  
  
  // DADA stuff
  
  syslog (LOG_INFO, "creating in and out hdus");
  
  hdu_in  = dada_hdu_create ();
  dada_hdu_set_key (hdu_in, in_key);
  if (dada_hdu_connect (hdu_in) < 0) {
    syslog (LOG_ERR,"could not connect to dada buffer in");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in) < 0) {
    syslog (LOG_ERR,"could not lock to dada buffer in");
    return EXIT_FAILURE;
  }

  hdu_out  = dada_hdu_create ();
  dada_hdu_set_key (hdu_out, out_key);
  if (dada_hdu_connect (hdu_out) < 0) {
    syslog (LOG_ERR,"could not connect to output  buffer");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out) < 0) {
    syslog (LOG_ERR, "could not lock to output buffer");
    return EXIT_FAILURE;
  }

  uint64_t header_size = 0;

  // deal with headers
  char * header_in = ipcbuf_get_next_read (hdu_in->header_block, &header_size);
  if (!header_in)
    {
      syslog(LOG_ERR, "could not read next header");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out);
      return EXIT_FAILURE;
    }
  if (ipcbuf_mark_cleared (hdu_in->header_block) < 0)
    {
      syslog (LOG_ERR, "could not mark header block cleared");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out);
      return EXIT_FAILURE;
    }

  char * header_out = ipcbuf_get_next_write (hdu_out->header_block);
  if (!header_out)
    {
      syslog(LOG_ERR, "could not get next header block [output]");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out);
      return EXIT_FAILURE;
    }
  memcpy (header_out, header_in, header_size);
  if (ipcbuf_mark_filled (hdu_out->header_block, header_size) < 0)
    {
      syslog (LOG_ERR, "could not mark header block filled [output]");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out);
      return EXIT_FAILURE;
    }

  // record STATE info
  sprintf(STATE,"LISTEN");
  syslog(LOG_INFO,"dealt with dada stuff - now in LISTEN state");  
  
  // get block sizes and allocate memory
  uint64_t block_size = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_in->data_block);
  uint64_t block_out = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_out->data_block);
  syslog(LOG_INFO, "main: have input and output block sizes %d %d\n",block_size,block_out);
  // check that these are correct
  // one time sample is NCHAN*NANT*NPOL bytes. Expect NREORDERS*NNATINTS*NCORRINTS times per block.
  if (block_size != NREORDERS*NNATINTS*NCORRINTS*NCHANG*NSNAPS*4608/2) {
    syslog(LOG_ERR,"wrong block_size %"PRIu64" in input",block_size);
    return EXIT_FAILURE;
  }
  // output is NBASE*NCHAN*NPOL*8 bytes (baseline, frequency, pol, r/i)
  if (block_out != NBASE*NCHAN*NPOL*8) {
    syslog(LOG_ERR,"wrong block_out %"PRIu64" in output",block_out);
    return EXIT_FAILURE;
  }
  

  uint64_t  bytes_read = 0;
  char * block;
  float * output_buffer;
  output_buffer = (float *)malloc(sizeof(float)*block_out/4);
  uint64_t written, block_id;  
  float * matr, * mati;
  matr = (float *)malloc(sizeof(float)*XGPU_SIZE);
  mati = (float *)malloc(sizeof(float)*XGPU_SIZE);

  // register input hdu with gpu
  dada_cuda_dbregister(hdu_in);

  // set up XGPU
  XGPUInfo xgpu_info;
  int syncOp = SYNCOP_DUMP;
  int xgpu_error = 0;
  xgpuInfo(&xgpu_info);
  XGPUContext context;
  context.array_h = NULL;
  context.matrix_h = NULL;
  xgpu_error = xgpuInit(&context, 0);
  if(xgpu_error) {
    syslog(LOG_ERR, "xGPU error %d", xgpu_error);
    dsaX_dbgpu_cleanup (hdu_in, hdu_out);
    return EXIT_FAILURE;
  }
  ComplexInput *array_h = context.array_h; // this is pinned memory
  Complex *cuda_matrix_h = context.matrix_h;
  float *output_vis = (float *)malloc(sizeof(float)*XGPU_SIZE);
  memset((char *)array_h,0,2*context.array_len);

  // set up reorder
  //char * ob, * ob_all;  
  //ob = (char *)malloc(sizeof(char)*block_feed);
  //ob_all = (char *)malloc(sizeof(char)*NNATINTS*NCORRINTS*NCHAN*32*2*2);
  //memset(ob_all,0,NNATINTS*NCORRINTS*NCHAN*32*2*2);
  thrust::device_vector<char> d_array(NNATINTS*NCORRINTS*NCHAN*NANT*2*2);
  thrust::fill(d_array.begin(),d_array.end(),0);
  thrust::device_vector<char> d_input(block_size);
  thrust::device_vector<char> d_output(block_feed);
  char *dinput = thrust::raw_pointer_cast(d_input.data());
  char *doutput = thrust::raw_pointer_cast(d_output.data());
  
  // get things started
  bool observation_complete=0;
  bool started = 0;
  syslog(LOG_INFO, "starting observation");
  int blocks = 0;

  
  while (!observation_complete) {

    if (DEBUG) syslog(LOG_DEBUG,"reading block");    
    
    // open block
    block = ipcio_open_block_read (hdu_in->data_block, &bytes_read, &block_id);

    if (started==0) {
      sprintf(STATE,"RUN");
      syslog(LOG_INFO,"now in RUN state");
      started=1;
    }    

    if (DEBUG) syslog(LOG_DEBUG,"reordering");    
    
    // DO STUFF

    // reorder block
    thrust::copy(block,block+block_size,d_input.begin());
    thrust::fill(d_output.begin(),d_output.end(),0);
    massage<<<512, 32>>>(dinput,doutput);
    hipDeviceSynchronize();
    //thrust::copy(d_output.begin(),d_output.end(),ob);    
    
    // zero matr and mati
    /*for (int i=0;i<XGPU_SIZE;i++) {
      matr[i] = 0.;
      mati[i] = 0.;
      } */   

    if (DEBUG) syslog(LOG_DEBUG,"looping over accums");    
    
    // loop over accumulations
    /*for (int accum=0;accum<NCORRINTS;accum++) {

      // get data
      thrust::copy(ob + accum*NNATINTS*NCHAN*NANT*NPOL, ob + (accum+1)*NNATINTS*NCHAN*NANT*NPOL,(char *)array_h);
    
      // run xGPU
      xgpu_error = xgpuCudaXengine(&context, syncOp);
      if(xgpu_error) {
	syslog(LOG_ERR, "xGPU error %d\n", xgpu_error);
	return EXIT_FAILURE;
      }

      // accumulate
      for (int i=0;i<XGPU_SIZE;i++) {
	matr[i] += cuda_matrix_h[i].real;
	mati[i] += cuda_matrix_h[i].imag;
	}
      
    }*/

    // NREORDERS*NNATINTS*NCORRINTS*NCHAN*NSNAPS*3*2*2;

    // loop over reorders, writing each time
    
    for (int reo=0;reo<NREORDERS;reo++) {

      // get data from d_output into array_h via scatter operation
      thrust::scatter(d_output.begin()+reo*NNATINTS*NCORRINTS*NCHAN*NSNAPS*3*2*2,d_output.begin()+(reo+1)*NNATINTS*NCORRINTS*NCHAN*NSNAPS*3*2*2,map.begin(),d_array.begin());     

      if (DEBUG) syslog(LOG_DEBUG,"running xGPU");
      
      // put it in array_h
      thrust::copy(d_array.begin(), d_array.end(), (char *)array_h);

      // run xGPU
      xgpu_error = xgpuCudaXengine(&context, syncOp);                                               
      if(xgpu_error) {
	syslog(LOG_ERR, "xGPU error %d\n", xgpu_error);                                               
	return EXIT_FAILURE;                                                                          
      }
    
      // accumulate
      for (int i=0;i<XGPU_SIZE;i++) {
	matr[i] = cuda_matrix_h[i].real;
	mati[i] = cuda_matrix_h[i].imag;
      }
    
      if (DEBUG) syslog(LOG_DEBUG,"extracting");    
    
      // simple extract
      simple_extract(matr,mati,output_buffer);
    
      // write to output
      written = ipcio_write (hdu_out->data_block, (char *)output_buffer, block_out);
      if (written < block_out)
	{
	  syslog(LOG_ERR, "main: failed to write all data to datablock [output]");
	  dsaX_dbgpu_cleanup (hdu_in, hdu_out);
	  return EXIT_FAILURE;
	}

      if (DEBUG) {
	syslog(LOG_DEBUG, "written block %d",blocks);
	blocks++;
      }

    }
      
    if (bytes_read < block_size)
      observation_complete = 1;

    ipcio_close_block_read (hdu_in->data_block, bytes_read);

  }


  free(output_buffer);
  free(matr);
  free(mati);
  //free(ob);
  //free(ob_all);
  dada_cuda_dbunregister(hdu_in);
  dsaX_dbgpu_cleanup (hdu_in, hdu_out);
  
}


