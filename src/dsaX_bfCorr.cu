#include "hip/hip_runtime.h"
// -*- c++ -*-
/* assumes input and output block size is appropriate - will seg fault otherwise*/
/*
Workflow is similar for BF and corr applications
 - copy data to GPU, convert to half-precision and calibrate while reordering
 - do matrix operations to populate large output vector
 */
#include <iostream>
#include <algorithm>
using std::cout;
using std::cerr;
using std::endl;
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <string.h>
#include <unistd.h>
#include <netdb.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netinet/in.h>
#include <time.h>
#include <syslog.h>
#include <pthread.h>

#include "dada_client.h"
#include "dada_def.h"
#include "dada_hdu.h"
#include "multilog.h"
#include "ipcio.h"
#include "ipcbuf.h"
#include "dada_affinity.h"
#include "ascii_header.h"
#include "dsaX_def.h"

#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

// required to prevent overflow in corr matrix multiply
#define halfFac 4

// beam sep
#define sep 1.0 // arcmin

/* global variables */
int DEBUG = 0;

// define structure that carries around device memory
typedef struct dmem {

  // initial data and streams
  char * h_input; // host input pointer
  char * d_input, * d_tx; // [NPACKETS_PER_BLOCK, NANTS, NCHAN_PER_PACKET, 2 times, 2 pol, 4-bit complex]
  
  // correlator pointers
  // giant array for r and i: [NCHAN_PER_PACKET, 2 pol, NANTS_PROCESS, NPACKETS_PER_BLOCK * 2 times]
  half * d_r, * d_i;
  // arrays for matrix multiply output: input [NANTS_PROCESS, NANTS_PROCESS]
  half * d_outr, *d_outi, *d_tx_outr, *d_tx_outi;
  // giant output array: [NBASE, NCHAN_PER_PACKET, 2 pol, 2 complex]
  float * d_output;
  
  // beamformer pointers
  char * d_big_input;
  half * d_br, * d_bi;
  half * weights_r, * weights_i; //weights: [arm, tactp, b]
  half * d_bigbeam_r, * d_bigbeam_i; //output: [tc, b]
  unsigned char * d_bigpower; //output: [b, tc]
  float * d_scf; // scale factor per beam
  float * d_chscf;
  float * h_winp;
  int * flagants, nflags;
  float * h_freqs, * d_freqs;

  // timing
  float cp, prep, cubl, outp;
  
} dmem;


// allocate device memory
void initialize(dmem * d, int bf) {
  
  // for correlator
  if (bf==0) {
    hipMalloc((void **)(&d->d_input), sizeof(char)*NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*2*2);
    hipMalloc((void **)(&d->d_r), sizeof(half)*NCHAN_PER_PACKET*2*NANTS*NPACKETS_PER_BLOCK*2);
    hipMalloc((void **)(&d->d_i), sizeof(half)*NCHAN_PER_PACKET*2*NANTS*NPACKETS_PER_BLOCK*2);
    hipMalloc((void **)(&d->d_tx), sizeof(char)*NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*2*2);
    hipMalloc((void **)(&d->d_output), sizeof(float)*NBASE*NCHAN_PER_PACKET*2*2);
    hipMalloc((void **)(&d->d_outr), sizeof(half)*NCHAN_PER_PACKET*2*2*NANTS*NANTS*halfFac);
    hipMalloc((void **)(&d->d_outi), sizeof(half)*NCHAN_PER_PACKET*2*2*NANTS*NANTS*halfFac);
    hipMalloc((void **)(&d->d_tx_outr), sizeof(half)*NCHAN_PER_PACKET*2*2*NANTS*NANTS*halfFac);
    hipMalloc((void **)(&d->d_tx_outi), sizeof(half)*NCHAN_PER_PACKET*2*2*NANTS*NANTS*halfFac);
  }

  // for beamformer
  if (bf==1) {
    hipMalloc((void **)(&d->d_input), sizeof(char)*(NPACKETS_PER_BLOCK)*(NANTS/2)*NCHAN_PER_PACKET*2*2);
    hipMalloc((void **)(&d->d_big_input), sizeof(char)*(NPACKETS_PER_BLOCK)*(NANTS)*NCHAN_PER_PACKET*2*2);
    hipMalloc((void **)(&d->d_tx), sizeof(char)*(NPACKETS_PER_BLOCK)*(NANTS/2)*NCHAN_PER_PACKET*2*2);
    hipMalloc((void **)(&d->d_br), sizeof(half)*NCHAN_PER_PACKET*2*(NANTS/2)*(NPACKETS_PER_BLOCK)*2);
    hipMalloc((void **)(&d->d_bi), sizeof(half)*NCHAN_PER_PACKET*2*(NANTS/2)*(NPACKETS_PER_BLOCK)*2);
    hipMalloc((void **)(&d->weights_r), sizeof(half)*2*4*(NANTS/2)*8*2*2*(NBEAMS/2)*(NCHAN_PER_PACKET/8));
    hipMalloc((void **)(&d->weights_i), sizeof(half)*2*4*(NANTS/2)*8*2*2*(NBEAMS/2)*(NCHAN_PER_PACKET/8));
    hipMalloc((void **)(&d->d_bigbeam_r), sizeof(half)*(NPACKETS_PER_BLOCK/4)*(NCHAN_PER_PACKET/8)*(NBEAMS/2));
    hipMalloc((void **)(&d->d_bigbeam_i), sizeof(half)*(NPACKETS_PER_BLOCK/4)*(NCHAN_PER_PACKET/8)*(NBEAMS/2));
    hipMalloc((void **)(&d->d_bigpower), sizeof(unsigned char)*(NPACKETS_PER_BLOCK/4)*(NCHAN_PER_PACKET/8)*(NBEAMS));
    hipMalloc((void **)(&d->d_scf), sizeof(float)*(NBEAMS/2)); // beam scale factor
    hipMalloc((void **)(&d->d_chscf), sizeof(float)*(NBEAMS/2)*(NCHAN_PER_PACKET/8)); // beam scale factor

    // input weights: first is [NANTS, E/N], then [NANTS, 48, 2pol, R/I]
    d->h_winp = (float *)malloc(sizeof(float)*(NANTS*2+NANTS*(NCHAN_PER_PACKET/8)*2*2));
    d->flagants = (int *)malloc(sizeof(int)*NANTS);
    d->h_freqs = (float *)malloc(sizeof(float)*(NCHAN_PER_PACKET/8));
    hipMalloc((void **)(&d->d_freqs), sizeof(float)*(NCHAN_PER_PACKET/8));

    // timers
    d->cp = 0.;
    d->prep = 0.;
    d->outp = 0.;
    d->cubl = 0.;
    
  }
  
}

// deallocate device memory
void deallocate(dmem * d, int bf) {

  hipFree(d->d_input);

  if (bf==0) {
    hipFree(d->d_r);
    hipFree(d->d_i);
    hipFree(d->d_tx);
    hipFree(d->d_output);
    hipFree(d->d_outr);
    hipFree(d->d_outi);
    hipFree(d->d_tx_outr);
    hipFree(d->d_tx_outi);
  }
  if (bf==1) {
    hipFree(d->d_tx);
    hipFree(d->d_br);
    hipFree(d->d_bi);
    hipFree(d->weights_r);
    hipFree(d->weights_i);
    hipFree(d->d_bigbeam_r);
    hipFree(d->d_bigbeam_i);
    hipFree(d->d_bigpower);
    hipFree(d->d_scf);
    hipFree(d->d_chscf);
    free(d->h_winp);
    free(d->flagants);
    hipFree(d->d_freqs);
    free(d->h_freqs);
  }
  
}

void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out);
int dada_bind_thread_to_core (int core);

void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out)
{

  if (dada_hdu_unlock_read (in) < 0)
    {
      syslog(LOG_ERR, "could not unlock read on hdu_in");
    }
  dada_hdu_destroy (in);

  if (dada_hdu_unlock_write (out) < 0)
    {
      syslog(LOG_ERR, "could not unlock write on hdu_out");
    }
  dada_hdu_destroy (out);

} 


void usage()
{
fprintf (stdout,
	 "dsaX_bfCorr [options]\n"
	 " -c core   bind process to CPU core [no default]\n"
	 " -d send debug messages to syslog\n"
	 " -i in_key [default REORDER_BLOCK_KEY]\n"
	 " -o out_key [default XGPU_BLOCK_KEY]\n"
	 " -b run beamformer [default is to run correlator]\n"
	 " -h print usage\n"
	 " -t binary file for test mode\n"
	 " -f flagants file\n"
	 " -a calib file\n"
	 " -s start frequency (assumes -0.244140625MHz BW)\n");
}

// kernel to fluff input
// run with 128 threads and NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*4/128 blocks
__global__ void corr_input_copy(char *input, half *inr, half *ini) {

  int bidx = blockIdx.x; // assume NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*4/128
  int tidx = threadIdx.x; // assume 128
  int iidx = bidx*128+tidx;
  
  inr[iidx] = __float2half((float)((char)(((unsigned char)(input[iidx]) & (unsigned char)(15)) << 4) >> 4));
  ini[iidx] = __float2half((float)((char)(((unsigned char)(input[iidx]) & (unsigned char)(240))) >> 4));

}


// arbitrary transpose kernel
// assume breakdown into tiles of 32x32, and run with 32x8 threads per block
// launch with dim3 dimBlock(32, 8) and dim3 dimGrid(Width/32, Height/32)
// here, width is the dimension of the fastest index
__global__ void transpose_matrix_char(char * idata, char * odata) {

  __shared__ char tile[32][33];
  
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 32 + threadIdx.y;
  int width = gridDim.x * 32;

  for (int j = 0; j < 32; j += 8)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * 32 + threadIdx.x;  // transpose block offset
  y = blockIdx.x * 32 + threadIdx.y;
  width = gridDim.y * 32;

  for (int j = 0; j < 32; j += 8)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];

}

// arbitrary transpose kernel
// assume breakdown into tiles of 32x32, and run with 32x8 threads per block
// launch with dim3 dimBlock(32, 8) and dim3 dimGrid(Width/32, Height/32)
// here, width is the dimension of the fastest index
__global__ void transpose_matrix_float(half * idata, half * odata) {

  __shared__ half tile[32][33];
  
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 32 + threadIdx.y;
  int width = gridDim.x * 32;

  for (int j = 0; j < 32; j += 8)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * 32 + threadIdx.x;  // transpose block offset
  y = blockIdx.x * 32 + threadIdx.y;
  width = gridDim.y * 32;

  for (int j = 0; j < 32; j += 8)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];

}


// function to copy amd reorder d_input to d_r and d_i
// input is [NPACKETS_PER_BLOCK, NANTS, NCHAN_PER_PACKET, 2 times, 2 pol, 4-bit complex]
// output is [NCHAN_PER_PACKET, 2times, 2pol, NPACKETS_PER_BLOCK, NANTS]
// starts by running transpose on [NPACKETS_PER_BLOCK * NANTS, NCHAN_PER_PACKET * 2 * 2] matrix in doubleComplex form.
// then fluffs using simple kernel
void reorder_input(char *input, char * tx, half *inr, half *ini) {

  // transpose input data
  dim3 dimBlock(32, 8), dimGrid((NCHAN_PER_PACKET*2*2)/32, ((NPACKETS_PER_BLOCK)*NANTS)/32);
  transpose_matrix_char<<<dimGrid,dimBlock>>>(input,tx);
  /*
  // set up for geam
  hipblasHandle_t cublasH = NULL;
  hipStream_t stream = NULL;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  hipblasSetStream(cublasH, stream);

  // transpose input matrix into tx
  hipblasOperation_t transa = HIPBLAS_OP_T;
  hipblasOperation_t transb = HIPBLAS_OP_N;
  const int m = NPACKETS_PER_BLOCK * NANTS;
  const int n = NCHAN_PER_PACKET*2*2/8; // columns in output
  const double alpha = 1.0;
  const double beta = 0.0;
  const int lda = n;
  const int ldb = m;
  const int ldc = ldb;
  hipblasDgeam(cublasH,transa,transb,m,n,
	      &alpha,(double *)(input),
	      lda,&beta,(double *)(tx),
	      ldb,(double *)(tx),ldc);
  */
  // now we just need to fluff to half-precision
  corr_input_copy<<<NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*4/128,128>>>(tx,inr,ini);

  // look at output
  /*char * odata = (char *)malloc(sizeof(char)*NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*4*2);
  hipMemcpy(odata,inr,NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*4*2,hipMemcpyDeviceToHost);
  FILE *fout;
  fout=fopen("test.test","wb");
  fwrite(odata,1,NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*4*2,fout);
  fclose(fout);*/
  
  // destroy stream
  //hipStreamDestroy(stream);
  
}

// kernel to help with reordering output
// outr and outi are [NANTS, NANTS, NCHAN_PER_PACKET, 2time, 2pol, halfFac]
// run with NCHAN_PER_PACKET*2*NBASE/128 blocks of 128 threads
__global__ void corr_output_copy(half *outr, half *outi, float *output, int *indices_lookup) {

  int bidx = blockIdx.x; // assume NCHAN_PER_PACKET*2*NBASE/128
  int tidx = threadIdx.x; // assume 128
  int idx = bidx*128+tidx;
  
  int baseline = (int)(idx / (NCHAN_PER_PACKET * 2));
  int chpol = (int)(idx % (NCHAN_PER_PACKET * 2));
  int ch = (int)(chpol / 2);
  int base_idx = indices_lookup[baseline];
  int iidx = base_idx * NCHAN_PER_PACKET + ch;
  int pol = (int)(chpol % 2);

  float v1=0., v2=0.;
  
  for (int i=0;i<halfFac;i++) {
    v1 += __half2float(outr[(4*iidx+pol)*halfFac+i])+__half2float(outr[(4*iidx+2+pol)*halfFac+i]);
    v2 += __half2float(outi[(4*iidx+pol)*halfFac+i])+__half2float(outi[(4*iidx+2+pol)*halfFac+i]);
  }

  output[2*idx] = v1;
  output[2*idx+1] = v2;
  
}


// function to copy d_outr and d_outi to d_output
// inputs are [NCHAN_PER_PACKET, 2 time, 2 pol, NANTS, NANTS]
// the corr matrices are column major order
// output needs to be [NBASE, NCHAN_PER_PACKET, 2 pol, 2 complex]
// start with transpose to get [NANTS*NANTS, NCHAN_PER_PACKET*2*2], then sum into output using kernel
void reorder_output(dmem * d) {

  // transpose input data
  dim3 dimBlock(32, 8), dimGrid((NANTS*NANTS)/32,(NCHAN_PER_PACKET*2*2*halfFac)/32);
  transpose_matrix_float<<<dimGrid,dimBlock>>>(d->d_outr,d->d_tx_outr);
  transpose_matrix_float<<<dimGrid,dimBlock>>>(d->d_outi,d->d_tx_outi);

  // look at output
  /*char * odata = (char *)malloc(sizeof(char)*384*4*NANTS*NANTS*2*halfFac);
  hipMemcpy(odata,d->d_tx_outr,384*4*NANTS*NANTS*2*halfFac,hipMemcpyDeviceToHost);
  FILE *fout;
  fout=fopen("test2.test","wb");
  fwrite(odata,sizeof(char),384*4*NANTS*NANTS*2*halfFac,fout);
  fclose(fout);*/

  
  /*
  // set up for geam
  hipblasHandle_t cublasH = NULL;
  hipStream_t stream = NULL;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  hipblasSetStream(cublasH, stream);

  // transpose output matrices into tx_outr and tx_outi
  hipblasOperation_t transa = HIPBLAS_OP_T;
  hipblasOperation_t transb = HIPBLAS_OP_N;
  const int m = NCHAN_PER_PACKET*2*2;
  const int n = NANTS*NANTS/16; // columns in output
  const double alpha = 1.0;
  const double beta = 0.0;
  const int lda = n;
  const int ldb = m;
  const int ldc = ldb;
  hipblasDgeam(cublasH,transa,transb,m,n,
	      &alpha,(double *)(d->d_outr),
	      lda,&beta,(double *)(d->d_tx_outr),
	      ldb,(double *)(d->d_tx_outr),ldc);
  hipblasDgeam(cublasH,transa,transb,m,n,
	      &alpha,(double *)(d->d_outi),
	      lda,&beta,(double *)(d->d_tx_outi),
	      ldb,(double *)(d->d_tx_outi),ldc);
  */
  // now run kernel to sum into output
  int * h_idxs = (int *)malloc(sizeof(int)*NBASE);
  int * d_idxs;
  hipMalloc((void **)(&d_idxs), sizeof(int)*NBASE);
  int ii = 0;
  // upper triangular order (column major) to match xGPU (not the same as CASA!)
  for (int i=0;i<NANTS;i++) {
    for (int j=0;j<=i;j++) {
      h_idxs[ii] = i*NANTS + j;
      ii++;
    }
  }
  hipMemcpy(d_idxs,h_idxs,sizeof(int)*NBASE,hipMemcpyHostToDevice);

  // run kernel to finish things
  corr_output_copy<<<NCHAN_PER_PACKET*2*NBASE/128,128>>>(d->d_tx_outr,d->d_tx_outi,d->d_output,d_idxs);

  /*char * odata = (char *)malloc(sizeof(char)*384*4*NBASE*4);
  hipMemcpy(odata,d->d_output,384*4*NBASE*4,hipMemcpyDeviceToHost);
  FILE *fout;
  fout=fopen("test3.test","wb");
  fwrite(odata,sizeof(char),384*4*NBASE*4,fout);
  fclose(fout);*/

  
  hipFree(d_idxs);
  free(h_idxs);
  //hipStreamDestroy(stream);  

}



// correlator function
// workflow: copy to device, reorder, stridedBatchedGemm, reorder
void dcorrelator(dmem * d) {

  // zero out output arrays
  hipMemset(d->d_outr,0,NCHAN_PER_PACKET*2*2*NANTS*NANTS*halfFac*sizeof(half));
  hipMemset(d->d_outi,0,NCHAN_PER_PACKET*2*2*NANTS*NANTS*halfFac*sizeof(half));
  hipMemset(d->d_output,0,NCHAN_PER_PACKET*2*NANTS*NANTS*sizeof(float));
  
  // copy to device
  hipMemcpy(d->d_input,d->h_input,NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*2*2,hipMemcpyHostToDevice);

  // reorder input
  reorder_input(d->d_input,d->d_tx,d->d_r,d->d_i);

  // not sure if essential
  hipDeviceSynchronize();
  
  // set up for gemm
  hipblasHandle_t cublasH = NULL;
  hipStream_t stream = NULL;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  hipblasCreate(&cublasH);
  hipblasSetStream(cublasH, stream);

  // gemm settings
  // input: [NCHAN_PER_PACKET, 2times, 2pol, NPACKETS_PER_BLOCK, NANTS]
  // output: [NCHAN_PER_PACKET, 2times, 2pol, NANTS, NANTS] 
  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_T;
  const int m = NANTS;
  const int n = NANTS;
  const int k = NPACKETS_PER_BLOCK/halfFac;
  const half alpha = 1.;
  const half malpha = -1.;
  const int lda = m;
  const int ldb = n;
  const half beta0 = 0.;
  const half beta1 = 1.;
  const int ldc = m;
  const long long int strideA = NPACKETS_PER_BLOCK*NANTS/halfFac;
  const long long int strideB = NPACKETS_PER_BLOCK*NANTS/halfFac;
  const long long int strideC = NANTS*NANTS;
  const int batchCount = NCHAN_PER_PACKET*2*2*halfFac;

  // run strided batched gemm
  // ac
  hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			    &alpha,d->d_r,lda,strideA,
			    d->d_r,ldb,strideB,&beta0,
			    d->d_outr,ldc,strideC,
			    batchCount);
  // bd
  hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			    &alpha,d->d_i,lda,strideA,
			    d->d_i,ldb,strideB,&beta1,
			    d->d_outr,ldc,strideC,
			    batchCount);
  // -bc
  hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			    &malpha,d->d_i,lda,strideA,
			    d->d_r,ldb,strideB,&beta0,
			    d->d_outi,ldc,strideC,
			    batchCount);
  // ad
  hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			    &alpha,d->d_r,lda,strideA,
			    d->d_i,ldb,strideB,&beta1,
			    d->d_outi,ldc,strideC,
			    batchCount);

  // shown to be essential
  hipDeviceSynchronize();

  // destroy stream
  hipStreamDestroy(stream);
  hipblasDestroy(cublasH);
  
  // reorder output data
  reorder_output(d);
  
}

// kernels to reorder and fluff input data for beamformer
// initial data is [NPACKETS_PER_BLOCK, (NANTS/2), NCHAN_PER_PACKET, 2 times, 2 pol, 4-bit complex]            
// want [NCHAN_PER_PACKET/8, NPACKETS_PER_BLOCK/4, 4tim, (NANTS/2), 8chan, 2 times, 2 pol, 4-bit complex]      // run as 16x16 tiled transpose with 32-byte words 
// launch with dim3 dimBlock(16, 8) and dim3 dimGrid(Width/16, Height/16)
// here, width=NCHAN_PER_PACKET/8 is the dimension of the fastest input index
// dim3 dimBlock1(16, 8), dimGrid1(NCHAN_PER_PACKET/8/16, (NPACKETS_PER_BLOCK)*(NANTS/2)/16);
__global__ void transpose_input_bf(double * idata, double * odata) {

  __shared__ double tile[16][17][4];
  
  int x = blockIdx.x * 16 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  int width = gridDim.x * 16;

  for (int j = 0; j < 16; j += 8) {
    tile[threadIdx.y+j][threadIdx.x][0] = idata[4*((y+j)*width + x)];
    tile[threadIdx.y+j][threadIdx.x][1] = idata[4*((y+j)*width + x)+1];
    tile[threadIdx.y+j][threadIdx.x][2] = idata[4*((y+j)*width + x)+2];
    tile[threadIdx.y+j][threadIdx.x][3] = idata[4*((y+j)*width + x)+3];
  }
  
  __syncthreads();

  x = blockIdx.y * 16 + threadIdx.x;  // transpose block offset
  y = blockIdx.x * 16 + threadIdx.y;
  width = gridDim.y * 16;

  for (int j = 0; j < 16; j += 8) {
    odata[4*((y+j)*width + x)] = tile[threadIdx.x][threadIdx.y + j][0];
    odata[4*((y+j)*width + x)+1] = tile[threadIdx.x][threadIdx.y + j][1];
    odata[4*((y+j)*width + x)+2] = tile[threadIdx.x][threadIdx.y + j][2];
    odata[4*((y+j)*width + x)+3] = tile[threadIdx.x][threadIdx.y + j][3];
  }

}

// kernel to fluff input bf data
// run with NPACKETS_PER_BLOCK*(NANTS/2)*NCHAN_PER_PACKET*2*2/128 blocks of 128 threads
__global__ void fluff_input_bf(char * input, half * dr, half * di) {

  int bidx = blockIdx.x; // assume NPACKETS_PER_BLOCK*(NANTS/2)*NCHAN_PER_PACKET*2*2/128
  int tidx = threadIdx.x; // assume 128
  int idx = bidx*128+tidx;

  dr[idx] = __float2half(0.015625*((float)((char)(((unsigned char)(input[idx]) & (unsigned char)(15)) << 4) >> 4)));
  di[idx] = __float2half(0.015625*((float)((char)(((unsigned char)(input[idx]) & (unsigned char)(240))) >> 4)));
  
}

// transpose, add and scale kernel for bf
// assume breakdown into tiles of 16x16, and run with 16x8 threads per block
// launch with dim3 dimBlock(16, 8) and dim3 dimGrid((NBEAMS/2)*(NPACKETS_PER_BLOCK/4)/16, (NCHAN_PER_PACKET/8)/16)
// scf is a per-beam scale factor to enable recasting as unsigned char
__global__ void transpose_scale_bf(half * ir, half * ii, unsigned char * odata) {

  __shared__ float tile[16][17];
  
  int x = blockIdx.x * 16 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  int width = gridDim.x * 16;
  float dr, di;

  for (int j = 0; j < 16; j += 8) {
    dr = (float)(ir[(y+j)*width + x]);
    di = (float)(ii[(y+j)*width + x]);
    tile[threadIdx.y+j][threadIdx.x] = (dr*dr+di*di);
  }

  __syncthreads();

  x = blockIdx.y * 16 + threadIdx.x;  // transpose block offset
  y = blockIdx.x * 16 + threadIdx.y;
  width = gridDim.y * 16;

  for (int j = 0; j < 16; j += 8)
    odata[(y+j)*width + x] = (unsigned char)(tile[threadIdx.x][threadIdx.y + j]/128.);

}

// sum over all times in output beam array
// run with (NCHAN_PER_PACKET/8)*(NBEAMS/2) blocks of (NPACKETS_PER_BLOCK/4) threads
__global__ void sum_beam(unsigned char * input, float * output) {

  __shared__ float summ[512];
  int bidx = blockIdx.x;
  int tidx = threadIdx.x;
  int idx = bidx*256+tidx;
  int bm = (int)(bidx/48);
  int ch = (int)(bidx % 48);

  summ[tidx] = (float)(input[bm*256*48 + tidx*48 + ch]);

  __syncthreads();

  if (tidx<256) {
    summ[tidx] += summ[tidx+256];
    summ[tidx] += summ[tidx+128];
    summ[tidx] += summ[tidx+64];
    summ[tidx] += summ[tidx+32];
    summ[tidx] += summ[tidx+16];
    summ[tidx] += summ[tidx+8];
    summ[tidx] += summ[tidx+4];
    summ[tidx] += summ[tidx+2];
    summ[tidx] += summ[tidx+1];
  }

  if (tidx==0) output[bidx] = summ[tidx];
  
}

/*
Beamformer:
 - initial data is [NPACKETS_PER_BLOCK, NANTS, NCHAN_PER_PACKET, 2 times, 2 pol, 4-bit complex] 
 - split into EW and NS antennas via hipMemcpy: [NPACKETS_PER_BLOCK, NANTS/2, NCHAN_PER_PACKET, 2 times, 2 pol, 4-bit complex]
 - want [NCHAN_PER_PACKET/8, NPACKETS_PER_BLOCK/4, 4tim, NANTS/2, 8chan, 2 times, 2 pol, 4-bit complex]
(single transpose operation)
 - weights are [NCHAN_PER_PACKET/8, NBEAMS, 4tim, NANTS/2, 8chan, 2 times, 2 pol] x 2
 - then fluff and run beamformer: output is [NCHAN_PER_PACKET/8, NBEAMS, NPACKETS_PER_BLOCK/4] (w column-major)
 - transpose and done! 

*/
// beamformer function
void dbeamformer(dmem * d) {

  // gemm settings - recall column major order assumed
  // stride over 48 chans
  hipblasHandle_t cublasH = NULL;
  hipblasCreate(&cublasH);
  hipblasOperation_t transa = HIPBLAS_OP_T;
  hipblasOperation_t transb = HIPBLAS_OP_N;
  const int m = NPACKETS_PER_BLOCK/4;
  const int n = NBEAMS/2;
  const int k = 4*(NANTS/2)*8*2*2;
  const half alpha = 1.;
  const half malpha = -1.;
  const int lda = k;
  const int ldb = k;
  const half beta0 = 0.;
  const half beta1 = 1.;
  const int ldc = m;
  const long long int strideA = (NPACKETS_PER_BLOCK)*(NANTS/2)*8*2*2;
  const long long int strideB = (NBEAMS/2)*4*(NANTS/2)*8*2*2;
  const long long int strideC = (NPACKETS_PER_BLOCK/4)*NBEAMS/2;
  const int batchCount = NCHAN_PER_PACKET/8;
  long long int i1, i2, o1;
  
  // create streams
  hipStream_t stream;
  hipStreamCreate(&stream);

  // timing
  // copy, prepare, cublas, output
  clock_t begin, end;

  // do big memcpy
  begin = clock();
  hipMemcpy(d->d_big_input,d->h_input,NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*4,hipMemcpyHostToDevice);
  end = clock();
  d->cp += (float)(end - begin) / CLOCKS_PER_SEC;
  
  // loop over halves of the array
  for (int iArm=0;iArm<2;iArm++) {
  
    // zero out output arrays
    hipMemset(d->d_bigbeam_r,0,(NPACKETS_PER_BLOCK/4)*(NCHAN_PER_PACKET/8)*(NBEAMS/2)*sizeof(half));
    hipMemset(d->d_bigbeam_i,0,(NPACKETS_PER_BLOCK/4)*(NCHAN_PER_PACKET/8)*(NBEAMS/2)*sizeof(half));
    hipDeviceSynchronize();
    
    // copy data to device
    // initial data: [NPACKETS_PER_BLOCK, NANTS, NCHAN_PER_PACKET, 2 times, 2 pol, 4-bit complex]
    // final data: need to split by NANTS.
    begin = clock();
    for (i1=0;i1<NPACKETS_PER_BLOCK;i1++) 
      hipMemcpy(d->d_input+i1*(NANTS/2)*NCHAN_PER_PACKET*4,d->d_big_input+i1*(NANTS)*NCHAN_PER_PACKET*4+iArm*(NANTS/2)*NCHAN_PER_PACKET*4,(NANTS/2)*NCHAN_PER_PACKET*4,hipMemcpyDeviceToDevice);
    end = clock();
    d->cp += (float)(end - begin) / CLOCKS_PER_SEC;
    
    // do reorder and fluff of data to real and imag
    begin = clock();
    dim3 dimBlock1(16, 8), dimGrid1(NCHAN_PER_PACKET/8/16, (NPACKETS_PER_BLOCK)*(NANTS/2)/16);
    transpose_input_bf<<<dimGrid1,dimBlock1>>>((double *)(d->d_input),(double *)(d->d_tx));
    fluff_input_bf<<<NPACKETS_PER_BLOCK*(NANTS/2)*NCHAN_PER_PACKET*2*2/128,128>>>(d->d_tx,d->d_br,d->d_bi);
    end = clock();
    d->prep += (float)(end - begin) / CLOCKS_PER_SEC;

    // large matrix multiply to get real and imag outputs
    // set up for gemm
    hipblasSetStream(cublasH, stream);
    i2 = iArm*4*(NANTS/2)*8*2*2*(NBEAMS/2)*(NCHAN_PER_PACKET/8); // weights offset
          
    // run strided batched gemm
    begin = clock();
    // ac
    hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			      &alpha,d->d_br,lda,strideA,
			      d->weights_r+i2,ldb,strideB,&beta0,
			      d->d_bigbeam_r,ldc,strideC,
			      batchCount);
    // -bd
    hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			      &malpha,d->d_bi,lda,strideA,
			      d->weights_i+i2,ldb,strideB,&beta1,
			      d->d_bigbeam_r,ldc,strideC,
			      batchCount);
    // bc
    hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			      &alpha,d->d_bi,lda,strideA,
			      d->weights_r+i2,ldb,strideB,&beta0,
			      d->d_bigbeam_i,ldc,strideC,
			      batchCount);
    // ad
    hipblasHgemmStridedBatched(cublasH,transa,transb,m,n,k,
			      &alpha,d->d_br,lda,strideA,
			      d->weights_i+i2,ldb,strideB,&beta1,
			      d->d_bigbeam_i,ldc,strideC,
			      batchCount);
      
    hipDeviceSynchronize();
    end = clock();
    d->cubl += (float)(end - begin) / CLOCKS_PER_SEC;
      
        
    // simple formation of total power and scaling to 8-bit in transpose kernel
    begin = clock();
    dim3 dimBlock(16, 8), dimGrid((NBEAMS/2)*(NPACKETS_PER_BLOCK/4)/16, (NCHAN_PER_PACKET/8)/16);
    transpose_scale_bf<<<dimGrid,dimBlock>>>(d->d_bigbeam_r,d->d_bigbeam_i,d->d_bigpower+iArm*(NPACKETS_PER_BLOCK/4)*(NCHAN_PER_PACKET/8)*(NBEAMS/2));
    end = clock();
    d->outp += (float)(end - begin) / CLOCKS_PER_SEC;
      

  }

  hipStreamDestroy(stream);


  hipblasDestroy(cublasH);

  // form sum over times
  //sum_beam<<<24576,512>>>(d->d_bigpower,d->d_chscf);
  
}

// kernel to populate an instance of weights matrix [2, (NCHAN_PER_PACKET/8), NBEAMS/2, 4times*(NANTS/2)*8chan*2tim*2pol]
// run with 2*(NCHAN_PER_PACKET/8)*(NBEAMS/2)*128*(NANTS/2)/128 blocks of 128 threads
__global__ void populate_weights_matrix(float * antpos_e, float * antpos_n, float * calibs, half * wr, half * wi, float * fqs) {

  int bidx = blockIdx.x;
  int tidx = threadIdx.x;
  int inidx = bidx*128+tidx;  
  
  // 2*(NCHAN_PER_PACKET/8)*(NBEAMS/2)*128*(NANTS/2)
  
  // get indices
  int iArm = (int)(inidx / ((NCHAN_PER_PACKET/8)*(NBEAMS/2)*128*(NANTS/2)));
  int iidx = (int)(inidx % ((NCHAN_PER_PACKET/8)*(NBEAMS/2)*128*(NANTS/2)));
  int fq = (int)(iidx / (128*(NANTS/2)*(NBEAMS/2)));
  int idx = (int)(iidx % (128*(NANTS/2)*(NBEAMS/2)));
  int bm = (int)(idx / (128*(NANTS/2)));
  int tactp = (int)(idx % (128*(NANTS/2)));
  int t = (int)(tactp / (32*(NANTS/2)));
  int actp = (int)(tactp % (32*(NANTS/2)));
  int a = (int)(actp / 32);
  int ctp = (int)(actp % 32);
  int c = (int)(ctp / 4);
  int tp = (int)(ctp % 4);
  int t2 = (int)(tp / 2);
  int pol = (int)(tp % 2);
  int widx = (a+48*iArm)*(NCHAN_PER_PACKET/8)*2*2 + fq*2*2 + pol*2;
  
  // calculate weights
  float theta, afac, twr, twi;
  if (iArm==0) {
    theta = sep*(127.-bm*1.)*PI/10800.; // radians
    afac = -2.*PI*fqs[fq]*theta/CVAC; // factor for rotate
    twr = cos(afac*antpos_e[a+48*iArm]);
    twi = sin(afac*antpos_e[a+48*iArm]);
    wr[inidx] = __float2half((twr*calibs[widx] - twi*calibs[widx+1]));
    wi[inidx] = __float2half((twi*calibs[widx] + twr*calibs[widx+1]));
    //wr[inidx] = __float2half(calibs[widx]);
    //wi[inidx] = __float2half(calibs[widx+1]);
  }
  if (iArm==1) {
    theta = sep*(127.-bm*1.)*PI/10800.; // radians
    afac = -2.*PI*fqs[fq]*theta/CVAC; // factor for rotate
    twr = cos(afac*antpos_n[a+48*iArm]);
    twi = sin(afac*antpos_n[a+48*iArm]);
    wr[inidx] = __float2half((twr*calibs[widx] - twi*calibs[widx+1]));
    wi[inidx] = __float2half((twi*calibs[widx] + twr*calibs[widx+1]));
    //wr[inidx] = __float2half(calibs[widx]);
    //wi[inidx] = __float2half(calibs[widx+1]);
  }
    
}

// GPU-powered function to populate weights matrix for beamformer
// file format:
// sequential pairs of eastings and northings
// then [NANTS, 48, R/I] calibs

void calc_weights(dmem * d) {

  // allocate
  float *antpos_e = (float *)malloc(sizeof(float)*NANTS);
  float *antpos_n = (float *)malloc(sizeof(float)*NANTS);
  float *calibs = (float *)malloc(sizeof(float)*NANTS*(NCHAN_PER_PACKET/8)*2*2);
  float *d_antpos_e, *d_antpos_n, *d_calibs;
  float wnorm;
  hipMalloc((void **)(&d_antpos_e), sizeof(float)*NANTS);
  hipMalloc((void **)(&d_antpos_n), sizeof(float)*NANTS);
  hipMalloc((void **)(&d_calibs), sizeof(float)*NANTS*(NCHAN_PER_PACKET/8)*2*2);

  // deal with antpos and calibs
  int iant, found;
  for (int i=0;i<NANTS;i++) {
    antpos_e[i] = d->h_winp[2*i];
    antpos_n[i] = d->h_winp[2*i+1];
  }
  for (int i=0;i<NANTS*(NCHAN_PER_PACKET/8)*2;i++) {

    iant = (int)(i/((NCHAN_PER_PACKET/8)*2));

    found = 0;
    for (int j=0;j<d->nflags;j++)
      if (d->flagants[j]==iant) found = 1;

    calibs[2*i] = d->h_winp[2*NANTS+2*i];
    calibs[2*i+1] = d->h_winp[2*NANTS+2*i+1];

    wnorm = sqrt(calibs[2*i]*calibs[2*i] + calibs[2*i+1]*calibs[2*i+1]);
    if (wnorm!=0.0) {
      calibs[2*i] /= wnorm;
      calibs[2*i+1] /= wnorm;
    }

    //if (found==1) {
    //calibs[2*i] = 0.;
    //calibs[2*i+1] = 0.;
    //}
  }

  //for (int i=0;i<NANTS*(NCHAN_PER_PACKET/8)*2;i++) printf("%f %f\n",calibs[2*i],calibs[2*i+1]);
  
  hipMemcpy(d_antpos_e,antpos_e,NANTS*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_antpos_n,antpos_n,NANTS*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_calibs,calibs,NANTS*(NCHAN_PER_PACKET/8)*2*2*sizeof(float),hipMemcpyHostToDevice);

  // run kernel to populate weights matrix
  populate_weights_matrix<<<2*(NCHAN_PER_PACKET/8)*(NBEAMS/2)*128*(NANTS/2)/128,128>>>(d_antpos_e,d_antpos_n,d_calibs,d->weights_r,d->weights_i,d->d_freqs);  
  
  // free stuff
  hipFree(d_antpos_e);
  hipFree(d_antpos_n);
  hipFree(d_calibs);
  free(antpos_e);
  free(antpos_n);
  free(calibs);
  
}

// MAIN

int main (int argc, char *argv[]) {

  hipSetDevice(1);
  
  // startup syslog message
  // using LOG_LOCAL0
  openlog ("dsaX_bfCorr", LOG_CONS | LOG_PID | LOG_NDELAY, LOG_LOCAL0);
  syslog (LOG_NOTICE, "Program started by User %d", getuid ());
  
  /* DADA Header plus Data Unit */
  dada_hdu_t* hdu_in = 0;
  dada_hdu_t* hdu_out = 0;

  // data block HDU keys
  key_t in_key = REORDER_BLOCK_KEY;
  key_t out_key = XGPU_BLOCK_KEY;
  
  // command line arguments
  int core = -1;
  int arg = 0;
  int bf = 0;
  int test = 0;
  char ftest[200], fflagants[200], fcalib[200];
  float sfreq = 1498.75;

  
  while ((arg=getopt(argc,argv,"c:i:o:t:f:a:s:bdh")) != -1)
    {
      switch (arg)
	{
	case 'c':
	  if (optarg)
	    {
	      core = atoi(optarg);
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-c flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'i':
	  if (optarg)
	    {
	      if (sscanf (optarg, "%x", &in_key) != 1) {
		syslog(LOG_ERR, "could not parse key from %s\n", optarg);
		return EXIT_FAILURE;
	      }
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-i flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'o':
	  if (optarg)
	    {
	      if (sscanf (optarg, "%x", &out_key) != 1) {
		syslog(LOG_ERR, "could not parse key from %s\n", optarg);
		return EXIT_FAILURE;
	      }
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-o flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 't':
	  if (optarg)
            {
	      test = 1;
	      syslog(LOG_INFO, "test mode");
	      if (sscanf (optarg, "%s", &ftest) != 1) {
		syslog(LOG_ERR, "could not read test file name from %s\n", optarg);
		return EXIT_FAILURE;
	      }
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-t flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'a':
	  if (optarg)
            {
	      syslog(LOG_INFO, "read calib file %s",optarg);
	      if (sscanf (optarg, "%s", &fcalib) != 1) {
		syslog(LOG_ERR, "could not read calib file name from %s\n", optarg);
		return EXIT_FAILURE;
	      }
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-a flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'f':
	  if (optarg)
            {
	      syslog(LOG_INFO, "reading flag ants file %s",optarg);
	      if (sscanf (optarg, "%s", &fflagants) != 1) {
		syslog(LOG_ERR, "could not read flagants file name from %s\n", optarg);
		return EXIT_FAILURE;
	      }
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-f flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 's':
	  if (optarg)
            {
	      sfreq = atof(optarg);
	      syslog(LOG_INFO, "start freq %g",sfreq);
 	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-s flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'd':
	  DEBUG=1;
	  syslog (LOG_DEBUG, "Will excrete all debug messages");
	  break;
	case 'b':
	  bf=1;
	  syslog (LOG_NOTICE, "Running beamformer, NOT correlator");
	  break;
	case 'h':
	  usage();
	  return EXIT_SUCCESS;
	}
    }

  // Bind to cpu core
  if (core >= 0)
    {
      if (dada_bind_thread_to_core(core) < 0)
	syslog(LOG_ERR,"failed to bind to core %d", core);
      syslog(LOG_NOTICE,"bound to core %d", core);
    }

  // allocate device memory
  dmem d;
  initialize(&d,bf);

  // set up for beamformer
  FILE *ff;
  int iii;
  if (bf) {

    if (!(ff=fopen(fflagants,"r"))) {
      syslog(LOG_ERR,"could not open flagants file\n");
      exit(1);
    }
    d.nflags=0;
    while (!feof(ff)) {
      fscanf(ff,"%d\n",&d.flagants[iii]);
      d.nflags++;
    }
    fclose(ff);

    if (!(ff=fopen(fcalib,"rb"))) {
      syslog(LOG_ERR,"could not open calibss file\n");
      exit(1);
    }
    fread(d.h_winp,NANTS*2+NANTS*(NCHAN_PER_PACKET/8)*2*2,4,ff);
    fclose(ff);

    for (iii=0;iii<(NCHAN_PER_PACKET/8);iii++)
      d.h_freqs[iii] = 1e6*(sfreq-iii*250./1024.);
    hipMemcpy(d.d_freqs,d.h_freqs,sizeof(float)*(NCHAN_PER_PACKET/8),hipMemcpyHostToDevice);

    // calculate weights
    calc_weights(&d);
    
  }

  // test mode
  FILE *fin, *fout;
  uint64_t output_size;
  char * output_data, * o1;
  if (test) {

    // read one block of input data    
    d.h_input = (char *)malloc(sizeof(char)*NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*2*2);
    for (int i=0;i<512;i++) {
      fin = fopen(ftest,"rb");
      fread(d.h_input+i*4*NANTS*NCHAN_PER_PACKET*2*2,4*NANTS*NCHAN_PER_PACKET*2*2,1,fin);
      fclose(fin);
    }

    // run correlator or beamformer, and output data
    if (bf==0) {
      if (DEBUG) syslog(LOG_INFO,"run correlator");
      dcorrelator(&d);
      if (DEBUG) syslog(LOG_INFO,"copy to host");
      output_size = NBASE*NCHAN_PER_PACKET*2*2*4;
      output_data = (char *)malloc(output_size);
      hipMemcpy(output_data,d.d_output,output_size,hipMemcpyDeviceToHost);

      fout = fopen("output.dat","wb");
      fwrite((float *)output_data,sizeof(float),NBASE*NCHAN_PER_PACKET*2*2,fout);
      fclose(fout);
    }
    else {
      if (DEBUG) syslog(LOG_INFO,"run beamformer");
      dbeamformer(&d);
      if (DEBUG) syslog(LOG_INFO,"copy to host");
      output_size = (NPACKETS_PER_BLOCK/4)*(NCHAN_PER_PACKET/8)*NBEAMS;
      output_data = (char *)malloc(output_size);
      hipMemcpy(output_data,d.d_bigpower,output_size,hipMemcpyDeviceToHost);

      /*output_size = 2*2*4*(NANTS/2)*8*2*2*(NBEAMS/2)*(NCHAN_PER_PACKET/8);
      o1 = (char *)malloc(output_size);
      hipMemcpy(o1,d.weights_r,output_size,hipMemcpyDeviceToHost);*/
	
      

      fout = fopen("output.dat","wb");
      fwrite((unsigned char *)output_data,sizeof(unsigned char),output_size,fout);
      //fwrite(o1,1,output_size,fout);
      fclose(fout);
    }

	
    // free
    free(d.h_input);
    free(output_data);
    free(o1);
    deallocate(&d,bf);

    exit(1);
  }
  


  
  // DADA stuff
  
  syslog (LOG_INFO, "creating in and out hdus");
  
  hdu_in  = dada_hdu_create ();
  dada_hdu_set_key (hdu_in, in_key);
  if (dada_hdu_connect (hdu_in) < 0) {
    syslog (LOG_ERR,"could not connect to dada buffer in");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in) < 0) {
    syslog (LOG_ERR,"could not lock to dada buffer in");
    return EXIT_FAILURE;
  }
  
  hdu_out  = dada_hdu_create ();
  dada_hdu_set_key (hdu_out, out_key);
  if (dada_hdu_connect (hdu_out) < 0) {
    syslog (LOG_ERR,"could not connect to output  buffer");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out) < 0) {
    syslog (LOG_ERR, "could not lock to output buffer");
    return EXIT_FAILURE;
  }

  uint64_t header_size = 0;

  // deal with headers
  char * header_in = ipcbuf_get_next_read (hdu_in->header_block, &header_size);
  if (!header_in)
    {
      syslog(LOG_ERR, "could not read next header");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out);
      return EXIT_FAILURE;
    }
  if (ipcbuf_mark_cleared (hdu_in->header_block) < 0)
    {
      syslog (LOG_ERR, "could not mark header block cleared");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out);
      return EXIT_FAILURE;
    }
  
  char * header_out = ipcbuf_get_next_write (hdu_out->header_block);
  if (!header_out)
    {
      syslog(LOG_ERR, "could not get next header block [output]");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out);
      return EXIT_FAILURE;
    }
  memcpy (header_out, header_in, header_size);
  if (ipcbuf_mark_filled (hdu_out->header_block, header_size) < 0)
    {
      syslog (LOG_ERR, "could not mark header block filled [output]");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out);
      return EXIT_FAILURE;
    }

  syslog(LOG_INFO,"dealt with dada stuff - now in LISTEN state");  
  
  // get block sizes and allocate memory
  uint64_t block_size = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_in->data_block);
  uint64_t block_out = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_out->data_block);
  syslog(LOG_INFO, "main: have input and output block sizes %d %d\n",block_size,block_out);
  if (bf==0) 
    syslog(LOG_INFO, "main: EXPECT input and output block sizes %d %d\n",NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*2*2,NBASE*NCHAN_PER_PACKET*2*2*4);
  else
    syslog(LOG_INFO, "main: EXPECT input and output block sizes %d %d\n",NPACKETS_PER_BLOCK*NANTS*NCHAN_PER_PACKET*2*2,(NPACKETS_PER_BLOCK/4)*(NCHAN_PER_PACKET/8)*NBEAMS);
  uint64_t  bytes_read = 0;
  char * block;
  char * output_buffer;
  output_buffer = (char *)malloc(block_out);
  uint64_t written, block_id;
  
  // get things started
  bool observation_complete=0;
  bool started = 0;
  syslog(LOG_INFO, "starting observation");
  int blocks = 0;
  clock_t begin, end;
  double time_spent;
  
  while (!observation_complete) {

    if (DEBUG) syslog(LOG_INFO,"reading block");    
    
    // open block
    d.h_input = ipcio_open_block_read (hdu_in->data_block, &bytes_read, &block_id);

    // do stuff
    //begin = clock();
    if (bf==0) {
      if (DEBUG) syslog(LOG_INFO,"run correlator");
      dcorrelator(&d);
      if (DEBUG) syslog(LOG_INFO,"copy to host");
      hipMemcpy(output_buffer,d.d_output,block_out,hipMemcpyDeviceToHost);
    }
    else {
      if (DEBUG) syslog(LOG_INFO,"run beamformer");
      dbeamformer(&d);
      if (DEBUG) syslog(LOG_INFO,"copy to host");
      hipMemcpy(output_buffer,d.d_bigpower,block_out,hipMemcpyDeviceToHost);
    }
    //end = clock();
    //time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    cout << "spent time " << d.cp << " " << d.prep << " " << d.cubl << " " << d.outp << " s" << endl;
    
    // write to output
    
    written = ipcio_write (hdu_out->data_block, (char *)(output_buffer), block_out);
    if (written < block_out)
      {
	syslog(LOG_ERR, "main: failed to write all data to datablock [output]");
	dsaX_dbgpu_cleanup (hdu_in, hdu_out);
	return EXIT_FAILURE;
      }
    
    if (DEBUG) syslog(LOG_INFO, "written block %d",blocks);	    
    blocks++;

    
      
    // finish up
    if (bytes_read < block_size)
      observation_complete = 1;

    ipcio_close_block_read (hdu_in->data_block, bytes_read);
    
  }

  // finish up
  free(output_buffer);
  deallocate(&d,bf);
  dsaX_dbgpu_cleanup (hdu_in, hdu_out);
  
}


